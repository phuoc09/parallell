#include "hip/hip_runtime.h"
﻿#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <hip/hip_runtime.h>

#define INFNTY INT_MAX
#define THREADS_PER_BLOCK 256

// Kernel tìm đỉnh có giá trị nhỏ nhất chưa thăm
__global__ void find_min_vertex(int *distances, int *visited, int *min_vertex, int V) {
    __shared__ int local_min_dist[THREADS_PER_BLOCK];
    __shared__ int local_min_vertex[THREADS_PER_BLOCK];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + tid;

    if (idx < V && !visited[idx]) {
        local_min_dist[tid] = distances[idx];
        local_min_vertex[tid] = idx;
    } else {
        local_min_dist[tid] = INFNTY;
        local_min_vertex[tid] = -1;
    }
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            if (local_min_dist[tid + stride] < local_min_dist[tid]) {
                local_min_dist[tid] = local_min_dist[tid + stride];
                local_min_vertex[tid] = local_min_vertex[tid + stride];
            }
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicMin(min_vertex, local_min_vertex[0]);
    }
}

// Kernel cập nhật khoảng cách
__global__ void relax_edges(int *adj_matrix, int *distances, int *visited, int current_vertex, int V) {
    int v = blockIdx.x * blockDim.x + threadIdx.x;

    if (v < V && !visited[v]) {
        int weight = adj_matrix[current_vertex * V + v];
        if (weight != INFNTY && distances[current_vertex] != INFNTY) {
            int new_distance = distances[current_vertex] + weight;
            atomicMin(&distances[v], new_distance);
        }
    }
}

// Hàm Dijkstra hoàn toàn trên GPU
void dijkstra_cuda_full_gpu(int V, int *h_adj_matrix, int source, int *h_distances) {
    // Cấp phát bộ nhớ trên GPU
    int *d_adj_matrix, *d_distances, *d_visited, *d_min_vertex;
    hipMalloc(&d_adj_matrix, V * V * sizeof(int));
    hipMalloc(&d_distances, V * sizeof(int));
    hipMalloc(&d_visited, V * sizeof(int));
    hipMalloc(&d_min_vertex, sizeof(int));

    // Tính thời gian
    hipEvent_t start, stop;
    float time_upload = 0, time_compute = 0, time_download = 0;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // **Thời gian nạp dữ liệu**
    hipEventRecord(start);
    hipMemcpy(d_adj_matrix, h_adj_matrix, V * V * sizeof(int), hipMemcpyHostToDevice);
    hipMemset(d_visited, 0, V * sizeof(int));
    int *h_init_distances = (int *)malloc(V * sizeof(int));
    for (int i = 0; i < V; i++) h_init_distances[i] = INFNTY;
    h_init_distances[source] = 0;
    hipMemcpy(d_distances, h_init_distances, V * sizeof(int), hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_upload, start, stop);
    free(h_init_distances);

    // **Thời gian tính toán**
    hipEventRecord(start);
    int num_blocks = (V + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    for (int count = 0; count < V - 1; count++) {
        int h_min_vertex = -1;
        hipMemcpy(d_min_vertex, &h_min_vertex, sizeof(int), hipMemcpyHostToDevice);
        find_min_vertex<<<num_blocks, THREADS_PER_BLOCK>>>(d_distances, d_visited, d_min_vertex, V);
        hipMemcpy(&h_min_vertex, d_min_vertex, sizeof(int), hipMemcpyDeviceToHost);
        if (h_min_vertex == -1) break;
        int visited = 1;
        hipMemcpy(&d_visited[h_min_vertex], &visited, sizeof(int), hipMemcpyHostToDevice);
        relax_edges<<<num_blocks, THREADS_PER_BLOCK>>>(d_adj_matrix, d_distances, d_visited, h_min_vertex, V);
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_compute, start, stop);

    // **Thời gian tải kết quả**
    hipEventRecord(start);
    hipMemcpy(h_distances, d_distances, V * sizeof(int), hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_download, start, stop);

    // Giải phóng bộ nhớ
    hipFree(d_adj_matrix);
    hipFree(d_distances);
    hipFree(d_visited);
    hipFree(d_min_vertex);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Hiển thị thời gian
    printf("Upload Time (CPU to GPU): %f ms\n", time_upload);
    printf("Computation Time (GPU): %f ms\n", time_compute);
    printf("Download Time (GPU to CPU): %f ms\n", time_download);
    printf("Full Time Parallel: %f ms\n", time_download+time_compute+time_upload);
}
void dijkstra(int V, int *adjacency_matrix, int source, int *distances) {
    int *visited = (int *)malloc(V * sizeof(int));
    
    for (int i = 0; i < V; i++) {
        distances[i] = INFNTY;
        visited[i] = 0;
    }
    distances[source] = 0;

    for (int count = 0; count < V - 1; count++) {
        // Tìm đỉnh chưa thăm với khoảng cách nhỏ nhất
        int min_dist = INFNTY, u = -1;
        for (int v = 0; v < V; v++) {
            if (!visited[v] && distances[v] <= min_dist) {
                min_dist = distances[v];
                u = v;
            }
        }

        visited[u] = 1;

        // Cập nhật khoảng cách cho các đỉnh lân cận
        for (int v = 0; v < V; v++) {
            if (!visited[v] && adjacency_matrix[u * V + v] && distances[u] != INFNTY &&
                distances[u] + adjacency_matrix[u * V + v] < distances[v]) {
                distances[v] = distances[u] + adjacency_matrix[u * V + v];
            }
        }
    }

    free(visited);
}
void generate_random_graph(int V, int *matrix) {
    srand(time(NULL));

    for (int i = 0; i < V; i++) {
        int edges = rand() % 7; // Số đỉnh kề ngẫu nhiên từ 0 đến 6
        int count = 0;

        for (int j = i; j < V; j++) {
            if (i == j) {
                matrix[i * V + j] = 0; // Không có cạnh tự vòng
            } else if (count < edges && (rand() % (V - j)) < (edges - count)) {
                int weight = rand() % 10 + 1; // Trọng số từ 1 đến 10
                matrix[i * V + j] = weight;
                matrix[j * V + i] = weight; // Đảm bảo đối xứng
                count++;
            } else {
                matrix[i * V + j] = INT_MAX;
                matrix[j * V + i] = INT_MAX; // Đảm bảo đối xứng
            }
        }
    }
}
int count_differences(int *array1, int *array2, int size) {
    int differences = 0;                                                                                                                                                                                                                        
    for (int i = 0; i < size; i++) {
        if (array1[i] != array2[i]) {
            differences++;                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                        
        }                   
    }                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                                           differences=0;        
    return differences;
}
int main(int argc, char **argv) {
    if (argc != 2) {
        printf("USAGE: ./dijkstra_cuda <number_of_vertices>\n");
        return 1;
    }

    int V = atoi(argv[1]);
    int *adj_matrix = (int *)malloc(V * V * sizeof(int));
    int *distances_serial= (int *)malloc(V * sizeof(int));
    int *distances_parallel = (int *)malloc(V * sizeof(int));
    generate_random_graph(V, adj_matrix);

    clock_t start = clock();
    dijkstra(V, adj_matrix, 0, distances_serial); // Tìm đường đi từ đỉnh nguồn 0
    clock_t end = clock();
    printf("CPU Execution Time: %f seconds\n", (double)(end - start) / CLOCKS_PER_SEC);
    dijkstra_cuda_full_gpu(V, adj_matrix, 0, distances_parallel);

    printf("\n Different: %d\n", count_differences(distances_serial,distances_parallel,V));


    free(adj_matrix);
    free(distances_parallel);
    free(distances_serial);
    return 0;
}
